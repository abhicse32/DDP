#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <string.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <thrust/device_vector.h>
#include "AVL_Concurrent_Lib.cu"
#include <>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ int getGlobalId(){
	int blockId= gridDim.x * gridDim.y * blockIdx.z + 
		     gridDim.x * blockIdx.y + blockIdx.x;
	int threadId= blockId*(blockDim.x * blockDim.y * blockDim.z)+ 
		      blockDim.x * blockDim.y * threadIdx.z + 
			blockDim.x * threadIdx.y + threadIdx.x;
	return threadId;
}

__global__ void start_kernel(Tree<int>* tree,int* d_query, int* max_query,
							int* three_4th, int* half, int* mutex){
	int threadId= getGlobalId();
	for(int i=threadId;i< *max_query;i+=512){
		if(i<*max_query){
			if(i<*half){
				tree->search_2(d_query[i]);
			}
			else{
				bool leave_loop=false;
				while(!leave_loop){
					if(atomicExch(mutex,1)==0){
						if(i<*three_4th)
							tree->delete_(d_query[i]);
						else tree->insert(d_query[i]);
						leave_loop=true;
						atomicExch(mutex,0);
					}
				}
			}
		}
	}
	__syncthreads();
}

int* generate_query(Tree<int>* tree,int size,int max_search){
	int* temp_arr= new int[max_search];
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC,&ts);
    srand((time_t)ts.tv_nsec);
    for(int i=0;i<max_search;++i){
        if(i%10==0){
        	int index= 2+rand()%(size-2);
	   	 	temp_arr[i]= tree->arr[index].key;
        }
        else
	    	temp_arr[i]= rand()%max_search;       
    }
    return temp_arr;
}

int main(int argc,char* argv[]){
	Tree<int>* tree= new Tree<int>();
	int max_search= atoi(argv[2]);
	tree->insert_nodes(argv[1],max_search);
 	
	int half= (max_search>>1);
	int three_4th=((3*max_search)>>2);
	int size_2= tree->size;
	// for(int i=0;i<5;++i){
 	int *d_query;
	int* d_max_query;
	Node<int>* d_arr;
	Tree<int>* d_tree;
   	int *d_half;
   	int *d_three_4th;
   	int *d_mutex;
    int*temp_arr= generate_query(tree,(size_2 - max_search),max_search);
    
	gpuErrchk(hipMalloc((void**)&d_arr,sizeof(Node<int>)*size_2));
	gpuErrchk(hipMalloc((void**)&d_tree,sizeof(Tree<int>)));
	gpuErrchk(hipMalloc((void**)&d_query,sizeof(int)*max_search));
	gpuErrchk(hipMalloc((void**)&d_max_query,sizeof(int)));
	gpuErrchk(hipMalloc((void**)&d_half,sizeof(int)));
	gpuErrchk(hipMalloc((void**)&d_mutex,sizeof(int)));
	gpuErrchk(hipMalloc((void**)&d_three_4th,sizeof(int)));

	gpuErrchk(hipMemcpy(d_tree,tree,sizeof(Tree<int>),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_arr,tree->arr,sizeof(Node<int>)*size_2,hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(&(d_tree->arr),&d_arr,sizeof(Node<int>*),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_query,temp_arr,sizeof(int)*max_search,hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_max_query,&max_search,sizeof(int),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_half,&half,sizeof(int),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_three_4th,&three_4th,sizeof(int),hipMemcpyHostToDevice));
	gpuErrchk(hipMemset(d_mutex,0,sizeof(int)));

	dim3 block_dim(8,8,8);
    dim3 grid_dim(8,8,8); 

	float elapsed_time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	start_kernel<<<grid_dim,block_dim>>>(d_tree,d_query,d_max_query, 
										d_three_4th,d_half,d_mutex);
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	// time_taken+=(elapsed_time/1000.0);

	hipFree(d_arr);
	hipFree(d_query);
	hipFree(d_tree);
	hipFree(d_half);
	hipFree(d_three_4th);
	hipFree(d_max_query);
	hipFree(d_mutex);
}
